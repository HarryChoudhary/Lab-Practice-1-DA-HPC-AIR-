#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

static const int WORK_SIZE = 10;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__global__ void sort(int *a,int i,int n)
{
	int tid = threadIdx.x;
	int p;
	int temp;
	if(i%2==0)
	{
		p=tid*2;

		if(a[p]>a[p+1])
		{
			temp = a[p];
			a[p] = a[p+1];
			a[p+1] =temp;
		}
	}
	else
	{
		p=tid*2+1;

		if(p<n-1)
		{
			if(a[p]>a[p+1])
			{
				temp = a[p];
				a[p] = a[p+1];
				a[p+1] =temp;
			}
		}
	}
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void)
{
	int a[WORK_SIZE];
	int i;
	int *da;

	CUDA_CHECK_RETURN(hipMalloc((void**) &da, sizeof(int) * WORK_SIZE));

	for(i=0;i<WORK_SIZE;i++)
	{
		printf("%d:",i);
		scanf("%d",&a[i]);
	}


	CUDA_CHECK_RETURN(hipMemcpy(da, a, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	for(i=0;i<WORK_SIZE;i++)
	{
		sort<<<1,WORK_SIZE/2>>>(da,i,WORK_SIZE);
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipMemcpy(a, da, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for(i=0;i<WORK_SIZE;i++)
	{
		printf("%d\t",a[i]);

	}

	printf("\n");

	CUDA_CHECK_RETURN(hipFree((void*) da));


	return 0;
}





Last login: Thu Mar 19 15:29:51 2015 from 10.80.0.68
echo $PWD'>'
/bin/sh -c "cd \"/tmp/nsight-debug\";\"/tmp/nsight-debug/OddEvenSort\"";exit
cuda-admin@cuda-admin:~$ echo $PWD'>'
/home/cuda-admin>
cuda-admin@cuda-admin:~$ /bin/sh -c "cd \"/tmp/nsight-debug\";\"/tmp/nsight-debu
g/OddEvenSort\"";exit
0 :
5
5

1 :
6
6

2 :
9
9

3 :
7
7

4 :
1
1

5 :
3
3

6 :
8
8

7 :
4
4

8 :
2
2

9 :
0
0
0 1 2 3 4 5 6 7 8 9

logout
