#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

static const int WORK_SIZE = 10;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
 /*
define CUDA_CHECK_RETURN(value) 
{											
	hipError_t _m_cudaStat = value;										
	if (_m_cudaStat != hipSuccess) {										
		fprintf(stderr, "Error %s at line %d in file %s\n",					
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);
		exit(1);															
	} }
*/
__global__ void sort(int *a,int i,int n)
{
	int tid = threadIdx.x;
	int p;
	int temp;
	if(i%2==0)
	{
		p=tid*2;

		if(a[p]>a[p+1])
		{
			temp = a[p];
			a[p] = a[p+1];
			a[p+1] =temp;
		}
	}
	else
	{
		p=tid*2+1;

		if(p<n-1)
		{
			if(a[p]>a[p+1])
			{
				temp = a[p];
				a[p] = a[p+1];
				a[p+1] =temp;
			}
		}
	}
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void)
{
	int a[WORK_SIZE];
	int i;
	int *da;

	hipMalloc((void**) &da, sizeof(int) * WORK_SIZE);

	for(i=0;i<WORK_SIZE;i++)
	{
		printf("%d:",i);
		scanf("%d",&a[i]);
	}


	hipMemcpy(da, a, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice);

	for(i=0;i<WORK_SIZE;i++)
	{
		sort<<<1,WORK_SIZE/2>>>(da,i,WORK_SIZE);
	}

	hipDeviceSynchronize();	// Wait for the GPU launched work to complete
	hipGetLastError();

	hipMemcpy(a, da, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost);

	for(i=0;i<WORK_SIZE;i++)
	{
		printf("%d\t",a[i]);

	}

	printf("\n");

	hipFree((void*) da);


	return 0;
}




