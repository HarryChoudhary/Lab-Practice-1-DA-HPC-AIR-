#include "stdio.h"
#include<hip/hip_runtime.h>
#define SOA 512

__global__ void vector_add(int *a,int *b,int *c){
	int id= threadIdx.x;
	c[id]=a[id]+b[id];
}

int main(void){

	int i;
	int *a,*b,*c;
	int *da,*db,*dc;
	int size=sizeof(int) * SOA;

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for(i=0;i<SOA;i++){
		a[i]= i;
		b[i]= i+1;
	}
	
	hipMalloc((void**)&da,size);
	hipMalloc((void**)&db,size);
	hipMalloc((void**)&dc,size);

	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	
	vector_add<<<1,SOA>>>(da,db,dc);

	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

	printf("Addition : ");
	for(i=0; i<SOA; i++)
	{
		printf("%d\n",c[i]);
	}
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;
	

}
	
