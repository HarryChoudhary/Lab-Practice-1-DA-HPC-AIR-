
#include <hip/hip_runtime.h>
#include<stdio.h>
#define Width 4
#define TILE_WIDTH 2
__global__ void mat_mul(int *a, int *b,int *ab, int width)
{
	// shorthand
	int tx = threadIdx.x, ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;
	// allocate tiles in __shared__ memory
	__shared__ int s_a[TILE_WIDTH][TILE_WIDTH];
	__shared__ int s_b[TILE_WIDTH][TILE_WIDTH];
	// calculate the row & col index
	int row = by*blockDim.y + ty;
	int col = bx*blockDim.x + tx;
	int result = 0;

	// loop over the tiles of the input in phases
	for(int p = 0; p < width/TILE_WIDTH; ++p)
	{
		// collaboratively load tiles into __shared__
		s_a[ty][tx] = a[row*width + (p*TILE_WIDTH + tx)];
		s_b[ty][tx] = b[(p*TILE_WIDTH + ty)*width + col];
		__syncthreads();
		// dot product between row of s_a and col of s_b
		for(int k = 0; k < TILE_WIDTH; ++k)
		result += s_a[ty][k] * s_b[k][tx];
		__syncthreads();
	}
	ab[row*width+col] = result;
}


int main()
{
    int mat_size=Width*Width*sizeof(int);	//Calculate memory size required for float matrix
    int tot_elements=Width*Width;
    int *M,*N,*P,*ptr;	// Host matrix pointers
	int a=0,x=1;
	int i=0;
	int *Md,*Nd,*Pd;		//Matrix Pointer on device memoryi.e GPU
	//int size=Width*Width*sizeof(int);
	

	M=(int*)malloc(mat_size);		//Allocate memory on host for matrix
	N=(int*)malloc(mat_size);
	P=(int*)malloc(mat_size);
	//P_CPU=(int*)malloc(mat_size);
	ptr=M;
	printf("\nGenarating random elements for matrix");
	for(i=0;i<tot_elements;i++)
	{	//a=(rand()%10);		//Generates random no. in 0 to 10 range
		//*ptr=a;
		*ptr=x++;
		ptr++;
	}
	ptr=N;
	for(i=0;i<tot_elements;i++)
	{
		//a=(rand()%10);
		*ptr=x--;
		ptr++;
	}
	printf("Matrix A=\n ");
	for(int i=0;i<Width*Width;i++)
	{	if(i%(Width)==0){
			printf("\n");
		}
		printf("%d ",M[i]);
	}
printf("Matrix B=\n ");
	for(int i=0;i<Width*Width;i++)
	{	if(i%(Width)==0){
			printf("\n");
		}
		printf("%d ",N[i]);
	}
	hipMalloc((void**)&Md,mat_size);		//Allocate memory on device global memory
	hipMemcpy(Md,M,mat_size,hipMemcpyHostToDevice);	//Copy matrix data from host to device memory
	hipMalloc((void**)&Nd,mat_size);
	hipMemcpy(Nd,N,mat_size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd,mat_size);

	dim3 dimGrid(TILE_WIDTH,TILE_WIDTH);	//Variable for threads arrangement in a block.
	dim3 dimBlock(Width/TILE_WIDTH,Width/TILE_WIDTH);		//Variable for blocks arrangement in a grid.	

	mat_mul<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);	//Kernel invocation with grid and block specification in angle brackets
	

	hipMemcpy(P,Pd,mat_size,hipMemcpyDeviceToHost);	//Copy resultant matrix from device to host
	//display the resultant matrix	
printf("Product=\n ");
	for(int i=0;i<Width*Width;i++)
	{	if(i%(Width)==0){
			printf("\n");
		}
		printf("%d ",P[i]);
	}
	//Free device memory
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
free(M);
free(N);
free(P);
}
