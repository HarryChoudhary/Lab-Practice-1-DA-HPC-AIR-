#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Thread block size
#define BLOCK_SIZE 64

//  Size of Array
#define SOA 512

// Allocates an array with random integer entries.
void randomInit(int* data, int size)
{
	for (int i = 0; i < size; ++i)
		data[i] = i+1;
}

__global__ void Sum(int *input, int *results, int n)    //take thread divergence into account
{
	__shared__ int sdata[BLOCK_SIZE];
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tx = threadIdx.x;
	 //load input into __shared__ memory
	int x = -INT_MAX;
	if(i < n)
		x = input[i];
	sdata[tx] = x;
	__syncthreads();

	// block-wide reduction
	for(unsigned int offset = blockDim.x>>1; offset > 0; offset >>= 1)
	{
		__syncthreads();
		if(tx < offset)
	    {
			sdata[tx]=sdata[tx]+sdata[tx+offset];
			//if(sdata[tx + offset] < sdata[tx])
			//	sdata[tx] = sdata[tx + offset];
		}

	}

		// finally, thread 0 writes the result
	if(threadIdx.x == 0)
	{
		// the result is per-block
		results[blockIdx.x] = sdata[0];
	}
}

// get global max element via per-block reductions
	int main()
	{
		int num_blocks = SOA / BLOCK_SIZE;
		int num_threads=BLOCK_SIZE,i;
		//allocate host memory for array a
		unsigned int mem_size_a = sizeof(int) * SOA;
		int* h_a = (int*)malloc(mem_size_a);

		//initialize host memory
		randomInit(h_a,SOA);

		//allocate device memory
		int* d_a;
		hipMalloc((void**) &d_a, mem_size_a);

		//copy host memory to device
		hipMemcpy(d_a, h_a, mem_size_a, hipMemcpyHostToDevice);

		//allocate device memory for temporary results
		unsigned int mem_size_b = sizeof(int) * num_blocks;
		int* d_b;
		hipMalloc((void**) &d_b, mem_size_b);
		int* h_b = (int*)malloc(mem_size_b);
		//allocate device memory for final result
		unsigned int mem_size_c = sizeof(int) ;
		int* d_c;
		hipMalloc((void**) &d_c, mem_size_c);

		//setup execution parameters
		//dim3 block(1,BLOCK_SIZE);
		//dim3 grid(4,4);

		//execute the kernel
		//first reduce per-block partial maxs
		Sum<<<num_blocks, num_threads>>>(d_a,d_b,SOA);
		hipMemcpy(h_b, d_b, mem_size_b, hipMemcpyDeviceToHost);
		//then reduce partial maxs to a final max
		Sum<<<1, num_blocks>>>(d_b,d_c,num_blocks);


       	// allocate host memory for the result
		int* h_c = (int*)malloc(mem_size_c);

		//copy final result from device to host
		hipMemcpy(h_c, d_c, mem_size_c, hipMemcpyDeviceToHost);

		double mean=*h_c/SOA;
		double *res=(double *)malloc(sizeof(double));
		for(int i=0;i<SOA;i++){
			*res=*res+((h_a[i]-mean)*(h_a[i]-mean));
		}
		double s=*res/SOA;
		double res1=pow(s,0.5);
		//print the result
		for(i=0;i<SOA;i++)
		{
		  printf("%d\t",h_a[i]);
		}
		printf("\n");
		for(i=0;i<num_blocks;i++)
		{
		  printf("%d\t",h_b[i]);
		}

		//print Final result
		printf("\nSum =%d\t",*h_c);

		printf("\nStandard deviation is= %f",res1);
		//clean up memory
		free(h_a);
		free(h_c);
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		hipDeviceReset();

	}
