#include "hip/hip_runtime.h"
Vector addition program

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define n 512

__global__ void bmk_add(int *a, int *b, int *result)
{
  int i = threadIdx.x;
  result[i] = a[i] + b[i];
}

int main()
{
  int num_blocks = 1, num_threads = n;

  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;

  int size = n * sizeof(int);

  a = (int*)malloc(size);
  b = (int*)malloc(size);
  c = (int*)malloc(size);

  hipMalloc((void**)&dev_a,size);
  hipMalloc((void**)&dev_b,size);
  hipMalloc((void**)&dev_c,size);

  for(int i = 0;i<n;i++)
  {
    //a[i] = rand()%1024;
    //b[i] = rand()%1024;
    a[i] = i;
    b[i] = i;
  }

  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

  bmk_add <<<num_blocks, num_threads>>>(dev_a,dev_b,dev_c);

  hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

  for(int i = 0;i<n;i++)
    printf("%d  ",c[i]);
    
    printf("\n");
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}

######################################################################################################################################################################################################
Multiplication program

#include<stdio.h>
#define Width 4
#define TILE_WIDTH 2
__global__ void mat_mul(int *a, int *b,int *ab, int width)
{
  // shorthand
  int tx = threadIdx.x, ty = threadIdx.y;
  int bx = blockIdx.x, by = blockIdx.y;
  // allocate tiles in __shared__ memory
  __shared__ int s_a[TILE_WIDTH][TILE_WIDTH];
  __shared__ int s_b[TILE_WIDTH][TILE_WIDTH];
  // calculate the row & col index
  int row = by*blockDim.y + ty;
  int col = bx*blockDim.x + tx;
  int result = 0;

  // loop over the tiles of the input in phases
  for(int p = 0; p < width/TILE_WIDTH; ++p)
  {
    // collaboratively load tiles into __shared__
    s_a[ty][tx] = a[row*width + (p*TILE_WIDTH + tx)];
    s_b[ty][tx] = b[(p*TILE_WIDTH + ty)*width + col];
    __syncthreads();
    // dot product between row of s_a and col of s_b
    for(int k = 0; k < TILE_WIDTH; ++k)
    result += s_a[ty][k] * s_b[k][tx];
    __syncthreads();
  }
  ab[row*width+col] = result;
}


int main()
{
    int mat_size=Width*Width*sizeof(int); //Calculate memory size required for float matrix
    int tot_elements=Width*Width;
    int *M,*N,*P,*ptr;  // Host matrix pointers
  int a=0,x=1;
  int i=0;
  int *Md,*Nd,*Pd;    //Matrix Pointer on device memoryi.e GPU
  //int size=Width*Width*sizeof(int);
  

  M=(int*)malloc(mat_size);   //Allocate memory on host for matrix
  N=(int*)malloc(mat_size);
  P=(int*)malloc(mat_size);
  //P_CPU=(int*)malloc(mat_size);
  ptr=M;
  printf("\nGenarating random elements for matrix");
  for(i=0;i<tot_elements;i++)
  { //a=(rand()%10);    //Generates random no. in 0 to 10 range
    //*ptr=a;
    *ptr=x++;
    ptr++;
  }
  ptr=N;
  for(i=0;i<tot_elements;i++)
  {
    //a=(rand()%10);
    *ptr=x--;
    ptr++;
  }
  printf("Matrix A=\n ");
  for(int i=0;i<Width*Width;i++)
  { if(i%(Width)==0){
      printf("\n");
    }
    printf("%d ",M[i]);
  }
printf("Matrix B=\n ");
  for(int i=0;i<Width*Width;i++)
  { if(i%(Width)==0){
      printf("\n");
    }
    printf("%d ",N[i]);
  }
  hipMalloc((void**)&Md,mat_size);   //Allocate memory on device global memory
  hipMemcpy(Md,M,mat_size,hipMemcpyHostToDevice); //Copy matrix data from host to device memory
  hipMalloc((void**)&Nd,mat_size);
  hipMemcpy(Nd,N,mat_size,hipMemcpyHostToDevice);
  hipMalloc((void**)&Pd,mat_size);

  dim3 dimGrid(TILE_WIDTH,TILE_WIDTH);  //Variable for threads arrangement in a block.
  dim3 dimBlock(Width/TILE_WIDTH,Width/TILE_WIDTH);   //Variable for blocks arrangement in a grid.  

  mat_mul<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);  //Kernel invocation with grid and block specification in angle brackets
  

  hipMemcpy(P,Pd,mat_size,hipMemcpyDeviceToHost); //Copy resultant matrix from device to host
  //display the resultant matrix  
printf("Product=\n ");
  for(int i=0;i<Width*Width;i++)
  { if(i%(Width)==0){
      printf("\n");
    }
    printf("%d ",P[i]);
  }
  //Free device memory
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
free(M);
free(N);
free(P);
}

