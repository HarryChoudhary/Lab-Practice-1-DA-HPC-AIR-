#include "hip/hip_runtime.h"
Maximum element program
#include<iostream>
using namespace std;

#define N 512
//Here one block multiple threads are used so(1,n/2) thread id is used instead of block id

__global__ void maxElement (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            if(a[tid] < a[tid+of])
            {
                a[tid] = a[tid+of];
            }
        }
    }

    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    maxElement<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    cout<<"Max Element is - "<<oh_a[0];

    hipFree(d_a);
    free(h_a);
}

######################################################################################################################################################################################################
Minimum element program


#include<iostream>
using namespace std;

#define N 512


__global__ void minElement (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            if(a[tid] > a[tid+of])
            {
                a[tid] = a[tid+of];
            }
        }
		syncthreads();
    }

    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    minElement<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    cout<<"Min Element is - "<<oh_a[0];

    hipFree(d_a);
    free(h_a);
}


######################################################################################################################################################################################################
Average program

#include<iostream>
using namespace std;

#define N 512


__global__ void ArithmeticMean (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            a[tid]+=a[tid+of];
        }
    }

    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }


    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    ArithmeticMean<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(h_a,d_a,size,hipMemcpyDeviceToHost);
    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);
    
    float AM =(float) oh_a[0]/N;
    cout<<"AM is "<<AM;

    hipFree(d_a);
    free(h_a);
}

######################################################################################################################################################################################################

Standard deviation program
#include<iostream>
using namespace std;

#define N 512


__global__ void Sum (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            a[tid]+=a[tid+of];
        }

    }

    o[0] = a[0];
}

__global__ void standardDeviation(int *a,int avg)
{
  int tid = threadIdx.x;
  if(tid<N)
  {
    a[tid] -= avg;
    a[tid] = a[tid]*a[tid];
  }
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a,*d_a1;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);
    //new
    hipMalloc((void**)&d_a1,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }

//sd=sqrt((sum(x-avg)^2)/n))
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_a1,h_a,size,hipMemcpyHostToDevice);

    Sum<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    int arithmetcMean = oh_a[0]/N;

    standardDeviation<<<1,N>>>(d_a1,arithmetcMean);

    Sum<<<1,N/2>>>(d_a1,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    int tmp = oh_a[0]/N;

    cout<<"Standard Deviation is - "<<sqrt(tmp)<<endl;

    hipFree(d_a);
    free(h_a);
    hipFree(o_a);
    free(oh_a);
    hipFree(d_a1);
    
}


